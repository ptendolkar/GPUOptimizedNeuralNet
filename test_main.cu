#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "dmatrix.h"
#include <stdio.h>

__global__ void newmatrix(Matrix * M, Matrix * N, Matrix *O )
{
	printf("inside kernel\n");
	M = new Matrix(3,3);
	M->print();
	printf("\n");
	M->initialize();

	M->print();
}

int main(int argc, char* argv[])
{
	int cuda_device = 0;
    	cuda_device = findCudaDevice(argc, (const char **)argv);

    	hipDeviceProp_t deviceProp;
    	checkCudaErrors(hipGetDevice(&cuda_device));
    	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * (1 << 20));
	
	std::cout << "before allocation" << std::endl;
	Matrix *d_M, *d_N, *d_O;
	hipMalloc(&d_M, sizeof(Matrix *));
	hipMalloc(&d_N, sizeof(Matrix *));
	hipMalloc(&d_O, sizeof(Matrix *));

	newmatrix<<<1,1>>>(d_M, d_N, d_O);
	hipDeviceSynchronize();

	std::cout << "completed" << std::endl;
		
	//saxpy(1.0, *d_M->begin(), 1, *d_N->begin(), 1);

	hipDeviceReset();
	return 0;

}
