#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
// CUDA runtime
#include <hip/hip_runtime.h>
//#include <hipblas.h>
// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dmatrix.h"

__global__ void newmatrix(Matrix *M)
{
	M = new Matrix(5,5);
}

int main(int argc, char* argv[])
{
	int cuda_device = 0;

    	cuda_device = findCudaDevice(argc, (const char **)argv);

    	hipDeviceProp_t deviceProp;
    	checkCudaErrors(hipGetDevice(&cuda_device));
    	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

	hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * (1 << 20));
	
	Matrix *d_M;
	hipMalloc(&d_M, sizeof(Matrix *));
	newmatrix<<<1,1>>>(d_M);
	
	std::cout << "completed" << std::endl;
		
	return 0;
}
