#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
// CUDA runtime
#include <hip/hip_runtime.h>
//#include <hipblas.h>
// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dmatrix.h"
#include<stdio.h>

__global__ void newmatrix(Matrix * M)
{
	printf("inside kernel\n");
	M = new Matrix(5,5);
	M->print();
	M->identity();
	M->print();
}

int main(int argc, char* argv[])
{
	int cuda_device = 0;

    	cuda_device = findCudaDevice(argc, (const char **)argv);

    	hipDeviceProp_t deviceProp;
    	checkCudaErrors(hipGetDevice(&cuda_device));
    	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * (1 << 20));
	
	std::cout << "before allocation" << std::endl;
	Matrix *d_M;
	hipMalloc(&d_M, sizeof(Matrix *));
	newmatrix<<<1,1>>>(d_M);
	hipDeviceSynchronize();

	std::cout << "completed" << std::endl;
		
	hipDeviceReset();
	return 0;
}
