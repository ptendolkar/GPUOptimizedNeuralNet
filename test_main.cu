#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
// CUDA runtime
#include <hip/hip_runtime.h>
//#include <hipblas.h>
// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "thrustMatrix.h"

double dtanh (double x) {
	return (1 - pow(tanh(x), 2));
}
double  sqloss(double x) {
	return 0.5*pow(x, 2);
}
double dsqloss(double x) {
	return x;
}
double lact(double x)
{
	return x;
}
double lgrd(double x)
{
	return 1.0;
}

/*__global__ void initializeMatrix(Matrix **M)
{

	}
*/
__global__ void newmatrix(Matrix *M){
	M = new Matrix(5,5);
}
int main(int argc, char* argv[])
{
	int cuda_device = 0;

    	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
    	cuda_device = findCudaDevice(argc, (const char **)argv);

    	hipDeviceProp_t deviceProp;
    	checkCudaErrors(hipGetDevice(&cuda_device));
    	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

	if (deviceProp.major < 2)
    	{
    	    printf("> This GPU with Compute Capability %d.%d does not meet minimum requirements.\n", deviceProp.major, deviceProp.minor);
    	    printf("> Test will not run.  Exiting.\n");
    	    exit(EXIT_SUCCESS);
    	}
	
	hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * (1 << 20));
	
	Matrix *d_M;
	hipMalloc(&d_M, sizeof(Matrix *));
	newmatrix<<<1,1>>>(d_M);
	
	std::cout << "completed" << std::endl;
		
	return 0;
}
