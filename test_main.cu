#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include "dneuron.h"

__global__ void train( Network *net, DevData *dd, float *dX, int  n_row, int n_col, int n_rsp, int n_fea)
{
	printf("in train kernel\n");
	for (int i= 0; i< n_row*n_col; i++){
		printf("%f ", dX[i]);
	}
	printf("\n");
	dd = new DevData(dX, n_row, n_col, n_rsp, n_fea);
}

int main(int argc, char* argv[])
{
	int cuda_device = 0;
    	cuda_device = findCudaDevice(argc, (const char **)argv);

    	hipDeviceProp_t deviceProp;
    	checkCudaErrors(hipGetDevice(&cuda_device));
    	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

	//hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * (1 << 20));
	
	std::cout << "before allocation" << std::endl;

	Data d("training", ' ', 2);
	DevData *dd;
	Network *net;
	
	hipMalloc(&dd, sizeof(DevData *));
	hipMalloc(&net, sizeof(Network *));

	train<<<1,1>>>(net, dd, thrust::raw_pointer_cast(&(d.X[0])), d.nrow(), d.ncol(), d.nrsp(), d.nfea());
	hipDeviceSynchronize();

	std::cout << "completed" << std::endl;
	hipDeviceReset();

	return 0;

}
